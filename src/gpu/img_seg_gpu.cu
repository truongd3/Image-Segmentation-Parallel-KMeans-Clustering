#include "hip/hip_runtime.h"
#include "common/constants.hpp"
#include "common/kmeans_utils.hpp"
#include "gpu/kmeans_kernels.cuh"
#include <cfloat>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <vector>

using namespace std;

#define CHECK_CUDA(err)                                                        \
    if (err != hipSuccess) {                                                  \
        cerr << "CUDA error " << hipGetErrorString(err) << " at "             \
                  << __LINE__ << '\n';                                         \
        exit(1);                                                               \
    }

namespace gpu {
void img_seg_gpu(size_t K, size_t N, const vector<float>& h_pixels,
                 vector<float>& h_centroids, vector<int>& h_labels) {

    kmeans_utils::init_centroids(h_pixels, h_centroids, N, K);

    float* d_pixels{};
    float* d_centroids{};
    float* d_sums{};
    int* d_labels{};
    int* d_counts{};

    const size_t d_pixels_size = N * PIXEL_DIM * sizeof(float);
    const size_t d_centroids_size = K * PIXEL_DIM * sizeof(float);
    const size_t d_labels_size = N * sizeof(int);
    const size_t d_sums_size = K * PIXEL_DIM * sizeof(float);
    const size_t d_counts_size = K * sizeof(int);

    CHECK_CUDA(hipMalloc(&d_pixels, d_pixels_size));
    CHECK_CUDA(hipMalloc(&d_centroids, d_centroids_size));
    CHECK_CUDA(hipMalloc(&d_labels, d_labels_size));
    CHECK_CUDA(hipMalloc(&d_sums, d_sums_size));
    CHECK_CUDA(hipMalloc(&d_counts, d_counts_size));

    CHECK_CUDA(hipMemcpy(d_pixels, h_pixels.data(), d_pixels_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_centroids, h_centroids.data(), d_centroids_size, hipMemcpyHostToDevice));

    dim3 blockDim(THREADS_PER_BLOCK);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    vector<float> h_sums(K * PIXEL_DIM);
    vector<int> h_counts(K);
    for (int iter = 0; iter < MAX_ITERS; iter++) {
        assign_clusters<<<gridDim, blockDim>>>(d_labels, d_pixels, d_centroids, N, K);

        CHECK_CUDA(hipDeviceSynchronize());

        CHECK_CUDA(hipMemset(d_sums, 0, d_sums_size));
        CHECK_CUDA(hipMemset(d_counts, 0, d_counts_size));

        accumulate_clusters<<<gridDim, blockDim>>>(d_pixels, d_labels, d_sums, d_counts, N);
        CHECK_CUDA(hipDeviceSynchronize());

        CHECK_CUDA(hipMemcpy(h_sums.data(), d_sums, d_sums_size, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_counts.data(), d_counts, d_counts_size, hipMemcpyDeviceToHost));

        // bool converged = true;
        for (int clus = 0; clus < K; clus++) {
            if (h_counts[clus] == 0) continue;
            for (int d = 0; d < PIXEL_DIM; d++) {
                float new_clus_comp = h_sums[(clus * PIXEL_DIM) + d] / (float)h_counts[clus];
                // if (abs(new_clus_comp - h_centroids[(clus * PIXEL_DIM) + d]) > TOL) converged = false;
                h_centroids[(clus * PIXEL_DIM) + d] = new_clus_comp;
            }
        }

        CHECK_CUDA(hipMemcpy(d_centroids, h_centroids.data(), d_centroids_size, hipMemcpyHostToDevice));

        // if (converged) {
        //     cout << "Converged at iteration " << iter << "\n";
        //     break;
        // }
    }
    CHECK_CUDA(hipMemcpy(h_labels.data(), d_labels, d_labels_size, hipMemcpyDeviceToHost));

    hipFree(d_pixels);
    hipFree(d_centroids);
    hipFree(d_labels);
    hipFree(d_sums);
    hipFree(d_counts);
}
} // namespace gpu

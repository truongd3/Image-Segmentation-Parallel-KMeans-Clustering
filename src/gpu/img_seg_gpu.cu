#include "hip/hip_runtime.h"
#include "common/constants.hpp"
#include "common/kmeans_utils.hpp"
#include <cfloat>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA(err)                                                        \
    if (err != hipSuccess) {                                                  \
        std::cerr << "CUDA error " << hipGetErrorString(err) << " at "        \
                  << __LINE__ << '\n';                                         \
        exit(1);                                                               \
    }

/**
 * @brief CUDA kernel to assign cluster for each data point
 * @param centroids array of centroids
 * @param labels store resulting cluster of each data point
 * @param N size of data points
 * @param K number of centroids
 */
__global__ void assign_clusters(int* labels, const float* pixels,
                                const float* centroids, size_t N, size_t K) {
    size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= N) {
        return;
    }

    float best_d = FLT_MAX;
    int best_k = 0;

    const float* pix = pixels + (idx * PIXEL_DIM);
    for (int c = 0; c < K; ++c) {
        const float* cent = centroids + (size_t)(c * PIXEL_DIM);
        float dist = 0;
        for (int d = 0; d < PIXEL_DIM; d++) {
            float diff = pix[d] - cent[d];
            dist += diff * diff;
        }

        if (dist < best_d) {
            best_d = dist;
            best_k = c;
        }
    }

    labels[idx] = best_k;
}

__global__ void accumulate_clusters(const float* pixels, const int* labels,
                                    float* sums, int* counts, size_t N) {
    size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= N) {
        return;
    }
    int clus = labels[idx];
    const float* pix = pixels + (idx * PIXEL_DIM);
    for (int d = 0; d < PIXEL_DIM; d++) {
        atomicAdd(&sums[(clus * PIXEL_DIM) + d], pix[d]);
    }
    atomicAdd(&counts[clus], 1);
}

namespace gpu {
void img_seg_gpu(size_t K, size_t N, const std::vector<float>& h_pixels,
                 std::vector<float>& h_centroids, std::vector<int>& h_labels) {

    kmeans_utils::init_centroids(h_pixels, h_centroids, N, K);

    float* d_pixels{};
    float* d_centroids{};
    float* d_sums{};
    int* d_labels{};
    int* d_counts{};

    const size_t d_pixels_size = N * PIXEL_DIM * sizeof(float);
    const size_t d_centroids_size = K * PIXEL_DIM * sizeof(float);
    const size_t d_labels_size = N * sizeof(int);
    const size_t d_sums_size = K * PIXEL_DIM * sizeof(float);
    const size_t d_counts_size = K * sizeof(int);

    CHECK_CUDA(hipMalloc(&d_pixels, d_pixels_size));
    CHECK_CUDA(hipMalloc(&d_centroids, d_centroids_size));
    CHECK_CUDA(hipMalloc(&d_labels, d_labels_size));
    CHECK_CUDA(hipMalloc(&d_sums, d_sums_size));
    CHECK_CUDA(hipMalloc(&d_counts, d_counts_size));

    CHECK_CUDA(hipMemcpy(d_pixels, h_pixels.data(), d_pixels_size,
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_centroids, h_centroids.data(), d_centroids_size,
                          hipMemcpyHostToDevice));

    dim3 blockDim(THREADS_PER_BLOCK);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    std::vector<float> h_sums(K * PIXEL_DIM);
    std::vector<int> h_counts(K);
    for (int iter = 0; iter < MAX_ITERS; ++iter) {
        assign_clusters<<<gridDim, blockDim>>>(d_labels, d_pixels, d_centroids,
                                               N, K);

        CHECK_CUDA(hipDeviceSynchronize());

        CHECK_CUDA(hipMemset(d_sums, 0, d_sums_size));
        CHECK_CUDA(hipMemset(d_counts, 0, d_counts_size));

        accumulate_clusters<<<gridDim, blockDim>>>(d_pixels, d_labels, d_sums,
                                                   d_counts, N);
        CHECK_CUDA(hipDeviceSynchronize());

        CHECK_CUDA(hipMemcpy(h_sums.data(), d_sums, d_sums_size,
                              hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_counts.data(), d_counts, d_counts_size,
                              hipMemcpyDeviceToHost));

        bool converged = true;
        for (int clus = 0; clus < K; ++clus) {
            if (h_counts[clus] == 0) {
                continue;
            }
            for (int d = 0; d < PIXEL_DIM; d++) {
                float new_clus_comp
                    = h_sums[(clus * PIXEL_DIM) + d] / (float)h_counts[clus];
                if (std::abs(new_clus_comp
                             - h_centroids[(clus * PIXEL_DIM) + d])
                    > TOL) {
                    converged = false;
                }
                h_centroids[(clus * PIXEL_DIM) + d] = new_clus_comp;
            }
        }

        CHECK_CUDA(hipMemcpy(d_centroids, h_centroids.data(), d_centroids_size,
                              hipMemcpyHostToDevice));

        if (converged) {
            std::cout << "Converged at iteration " << iter << "\n";
            break;
        }
    }
    CHECK_CUDA(hipMemcpy(h_labels.data(), d_labels, d_labels_size,
                          hipMemcpyDeviceToHost));

    hipFree(d_pixels);
    hipFree(d_centroids);
    hipFree(d_labels);
    hipFree(d_sums);
    hipFree(d_counts);
}
} // namespace gpu
